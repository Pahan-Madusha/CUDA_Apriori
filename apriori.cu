
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <utility>
#include <cstdlib>

__constant__ unsigned int d_lookup[256];

int get_one(unsigned int value);

struct MyBitMap {
	unsigned int *bits;
	int x,y;
	unsigned long long int size;

	MyBitMap(int row, int col) {
		int intCols = (col+31)/32;
		size = (unsigned long long int)row*(unsigned long long int)intCols;
		//printf("Need size: %llu, row: %d, cols: %d \n", size, row, intCols);
		bits = new unsigned int[size];
		x = row;
		y = intCols;
		for(int i=0; i<row*intCols; i++)
			bits[i] = 0;
	}
	~MyBitMap() {
		delete [] bits;
	}
	int getSize(){
		return x*y;
	}
	unsigned int *getPointer() {
		return bits;
	}
	int getRow() {
		return x;
	}
	int getCol() {
		return y;
	}
	void setRow(int row1, unsigned int *second, int row2) {
		for(int i=0; i<y; i++) {
			bits[row1*y+i] = second[row2*y+i];
		}
	}
	void resize(int row, int col) {
		delete [] bits;
		int intCols = (col+31)/32;
		size = (unsigned long long int)row*(unsigned long long int)intCols;
		//printf("Need size: %llu \n", size);
		bits = new unsigned int[size];
		x = row;
		y = intCols;
		for(int i=0; i<row*intCols; i++)
			bits[i] = 0;
	}
	unsigned int getInt(int row, int colInt) {
		return bits[row*y+colInt];
	}
	void setInt(int row, int colInt, unsigned int value) {
		bits[row*y+colInt] = value;
	}
	void setBit(int row, int col) {
		int i = row*y+col/32;
		unsigned int flag = 1;
		flag = flag<<(31-col%32);
		bits[i] = bits[i] | flag;
	}
	void clearBit(int row, int col) {
		int i = row*y+col/32;	
		unsigned int flag = 1;
		flag = flag<<(31-col%32);
		if((bits[i]&flag) != 0)
			bits[i] = bits[i] - flag;
	}
	unsigned int getBit(int row, int col) {
		int i = row*y+col/32;
		unsigned int flag = 1;
		flag = flag<<(31-col%32);
		if((flag&bits[i]) == 0)
			return 0;
		else
			return 1;	
	}
	void print(int row) {
		for(int i=0; i<y; i++)
			std::cout<<bits[row*y+i]<<" ";
	}
};

__global__ void count_ones(unsigned int *d_itemBitmap, unsigned int *d_bitmap, int numItem, int numTxn, int support)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	for (int i=idx; i<numItem; i += blockDim.x*gridDim.x) {
		int count = 0;
		int colInt = (numTxn+31)/32;
		for(int j=0; j<colInt; ++j){
			unsigned int temp = d_bitmap[i*colInt+j];
			unsigned int one = 255;
			one = one&temp;
			temp=temp>>8;
			unsigned int two = 255;
			two = two&temp;
			temp=temp>>8;
			unsigned int three = 255;
			three = three&temp;
			unsigned int four = temp>>8; 
			count += d_lookup[one]+d_lookup[two]+d_lookup[three]+d_lookup[four];
		}
		if(count >= support){
			int itemMapCol = (numItem+1+32)/32;	
			int index = itemMapCol*i+itemMapCol-1;
			unsigned int flag = 1;
			flag = flag<<(31-numItem%32);
			d_itemBitmap[index] = d_itemBitmap[index] | flag;
		}
	}
}

__global__ void testSupport(unsigned int *pairs, unsigned int *d_parent_transactions, unsigned int *d_child_transactions, unsigned int *d_child_items, int numItem, int support, int numTxn, int numChild)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	for (int i=idx; i<numChild; i += blockDim.x*gridDim.x) {
		int count = 0;
		int colTxn = (numTxn+31)/32;
		int colItem = (numItem+32)/32;
		for(int j=0; j<colTxn; ++j) {
			int a = pairs[2*i];
			int b = pairs[2*i+1];
			unsigned int temp = d_parent_transactions[a*colTxn+j] & d_parent_transactions[b*colTxn+j];
			d_child_transactions[i*colTxn+j]=temp;
			
			unsigned int one = 255;
			one = one&temp;
			temp=temp>>8;
			unsigned int two = 255;
			two = two&temp;
			temp=temp>>8;
			unsigned int three = 255;
			three = three&temp;
			unsigned int four = temp>>8; 
			count += d_lookup[one]+d_lookup[two]+d_lookup[three]+d_lookup[four];
		}
		if(count >= support) {
			int indexHere = colItem*(i+1)-1; 			
			unsigned int flag=1;
			flag = flag<<(31-numItem%32);
			d_child_items[indexHere] = d_child_items[indexHere] | flag;
		}
	}
}

__global__ void generateNext(unsigned int *pairs, unsigned int *d_parent, unsigned int *d_child, int itemSize, int itemNum, int size, int rowsItem)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	for (int i=idx; i<size; i += blockDim.x*gridDim.x) {
		int a=0;
		int b;
		int newI = i+1;
		int temp = rowsItem-1;
		while(newI>temp) {
			a++;
			newI -= temp;
			temp--;
		}
		b=a+newI;
		int colInt = (itemNum+32)/32;
		int equal = itemSize-2;
		for(int p=0; p<colInt; p++) {
			unsigned int aParent = d_parent[a*colInt+p];
			unsigned int bParent = d_parent[b*colInt+p];
			//printf("a: %d, b: %d, avalue: %u, bvalue: %u, p: %d, equal: %d\n",a,b, aParent, bParent, p, equal);
			unsigned int flag = 1;
			flag = flag<<31;
			int satisfy=1;
			for(int q=0; q<32; q++) {
				if(equal==0) {
					satisfy = 2;
					break;
				}
				if((aParent&flag) != (bParent&flag)){
					satisfy = 0;
					break;
				}
				else {
					if((aParent&flag)!=0)
						--equal;
				}
				flag = flag>>1;
			}
			if(satisfy==2) {
				for(int m=0; m<colInt; m++){
					unsigned int aNewParent = d_parent[a*colInt+m];
					unsigned int bNewParent = d_parent[b*colInt+m];
					d_child[i*colInt+m] = aNewParent | bNewParent;
				}
				int indexHere = (i+1)*colInt-1;
				unsigned int flag=1;
				flag = flag<<(31-itemNum%32);	
				d_child[indexHere] = d_child[indexHere] | flag;	
				pairs[i*2] = a;
				pairs[i*2+1] = b;
				//printf("satisfied a: %d, b: %d , d_childlast: %u \n",a, b, d_child[indexHere]);
				break;
			}
			if(satisfy==0){
				int indexHere = (i+1)*colInt-1;
				d_child[indexHere] = 0;	
				break;
			}
		}
	}
}

int main(int argc, char *argv[])
{
	std::ifstream input_file(argv[1]);
	int numBlock = atoi(argv[2]);
	int numThreads = atoi(argv[3]);
	float support_ratio=0.01;
	int tnx, numItem;
	input_file>>tnx>>numItem;
	float totalTime = 0;
	MyBitMap bitmap(numItem, tnx);
	int support = tnx*support_ratio;
	std::string tempLine;
	std::getline(input_file, tempLine);
	for(int i=0; i<tnx; i++) {
		std::string oneline;
		std::getline(input_file, oneline);
		std::istringstream items(oneline);
		int item;
		while(items>>item){	
			if (item<=numItem && item >0)
				bitmap.setBit(item-1, i);	
		}
		items.clear();
	}
	MyBitMap itemBitmap(numItem, numItem+1);
	for(int i=0; i<numItem; i++) {
		itemBitmap.setBit(i, i);
	}
		
	int lookup[256];
	for(unsigned int i=0; i<256; i++) {
		lookup[i]=get_one(i);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(d_lookup), lookup, sizeof(int)*256);
	unsigned int *d_bitmap, *d_itemBitmap;
	hipMalloc(&d_bitmap, bitmap.getSize()*sizeof(unsigned int));
	hipMalloc(&d_itemBitmap, itemBitmap.getSize()*sizeof(unsigned int));

	hipMemcpy(d_bitmap, bitmap.getPointer(), bitmap.getSize()*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_itemBitmap, itemBitmap.getPointer(), itemBitmap.getSize()*sizeof(unsigned int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	count_ones<<<numBlock, numThreads>>>(d_itemBitmap, d_bitmap, numItem, tnx, support);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime+=milliseconds;
	std::cout<<"Init time: "<<milliseconds<<"--------------------------"<<std::endl;

	hipMemcpy(bitmap.getPointer(),d_bitmap, bitmap.getSize()*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(itemBitmap.getPointer(), d_itemBitmap, itemBitmap.getSize()*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipFree(d_bitmap);
	hipFree(d_itemBitmap);
	int oldCount = numItem;
	int newCount = 0;	
	for(int i=0; i<numItem; i++) {
		if(itemBitmap.getBit(i, numItem) == 1)
			newCount++;
	}
	
	int tnxCol = (tnx+31)/32;
	int itemCol = (numItem+32)/32;
	int itemSize = 1;
	while(newCount > 1) {
		std::cout<<std::endl<<"new itemSize: "<<itemSize<<"  newCount: "<<newCount<<std::endl<<std::endl;
		itemSize++;
		MyBitMap newBitmap(newCount, tnx);
		MyBitMap newItemmap(newCount, numItem+1);
		int j=0;
		for(int i=0; i<oldCount; i++) {
			if(itemBitmap.getBit(i, numItem) == 1) {
				newBitmap.setRow(j, bitmap.getPointer(), i);	
				newItemmap.setRow(j, itemBitmap.getPointer(), i);
				newItemmap.clearBit(j, numItem);
				j++;
			}
		}
		int possibleNextChild = (newCount)*(newCount-1)/2;	
		unsigned int *d_pairs, *d_parent, *d_child;
		hipMalloc(&d_pairs, 2*possibleNextChild*sizeof(unsigned int));		
		hipMalloc(&d_parent, newCount*sizeof(unsigned int)*itemCol);
		hipMalloc(&d_child, possibleNextChild*itemCol*sizeof(unsigned int));
		printf("Device Variable alloc:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipMemcpy(d_parent, newItemmap.getPointer(), newItemmap.getSize()*sizeof(unsigned int), hipMemcpyHostToDevice);
		hipEventRecord(start);
		generateNext<<<numBlock, numThreads>>> (d_pairs, d_parent, d_child, itemSize, numItem, possibleNextChild, newCount);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		totalTime+=milliseconds;
		std::cout<<"generate time: "<<milliseconds<<"--------------------------"<<std::endl;
		unsigned int *pairs = new unsigned int[2*possibleNextChild];
		MyBitMap child(possibleNextChild, numItem+1);
		hipError_t error1 = hipMemcpy(pairs, d_pairs, 2*possibleNextChild*sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipError_t error2 = hipMemcpy(child.getPointer(), d_child, itemCol*possibleNextChild*sizeof(unsigned int), hipMemcpyDeviceToHost);
		//printf("Error1: %s\n", cudaGetErrorString(error1));
		//printf("Error2: %s\n", cudaGetErrorString(error2));
		printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipFree(d_child);
		hipFree(d_pairs);
		hipFree(d_parent);
		int usefulChild=0;
		for(int m=0; m<possibleNextChild; m++) {
			if(child.getBit(m,numItem) == 1)
				usefulChild++;
		}
		unsigned int *pairsGen = new unsigned int[2*usefulChild];
		std::cout<<std::endl<<"usefulChild:"<<usefulChild<<std::endl<<std::endl;
		itemBitmap.resize(usefulChild, numItem+1);
		j=0;
		for(int m=0; m<possibleNextChild; m++) {
			if(child.getBit(m, numItem) == 1) {
				itemBitmap.setRow(j, child.getPointer(), m);
				itemBitmap.clearBit(j, numItem);
				pairsGen[j*2]=pairs[2*m];
				pairsGen[j*2+1]=pairs[2*m+1];
				++j;
			}
		}
		delete []pairs;
		unsigned int *d_parent_tnx, *d_child_tnx, *d_child_item;
		hipMalloc(&d_pairs, 2*usefulChild*sizeof(unsigned int));		
		hipMalloc(&d_parent_tnx, newCount*sizeof(unsigned int)*tnxCol);		
		hipMalloc(&d_child_tnx, usefulChild*sizeof(unsigned int)*tnxCol);		
		hipMalloc(&d_child_item, usefulChild*sizeof(unsigned int)*itemCol);
		hipMemcpy(d_pairs, pairsGen, 2*usefulChild*sizeof(unsigned int),hipMemcpyHostToDevice);
		hipMemcpy(d_parent_tnx,newBitmap.getPointer() , newCount*sizeof(unsigned int)*tnxCol,hipMemcpyHostToDevice);
		hipMemcpy(d_child_item,itemBitmap.getPointer() , usefulChild*sizeof(unsigned int)*itemCol,hipMemcpyHostToDevice);
		hipEventRecord(start);
		testSupport<<<numBlock, numThreads>>> (d_pairs, d_parent_tnx, d_child_tnx, d_child_item, numItem, support, tnx, usefulChild);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		totalTime+=milliseconds;
		std::cout<<"test time: "<<milliseconds<<"--------------------------"<<std::endl;
		bitmap.resize(usefulChild, tnx);
		hipMemcpy(itemBitmap.getPointer(), d_child_item, usefulChild*sizeof(unsigned int)*itemCol, hipMemcpyDeviceToHost);
		hipMemcpy(bitmap.getPointer(), d_child_tnx, usefulChild*sizeof(unsigned int)*tnxCol, hipMemcpyDeviceToHost);
		newCount = 0;
		for(int m=0; m<usefulChild; m++) {
			if(itemBitmap.getBit(m, numItem) == 1)
				newCount++;
		}
		oldCount = usefulChild;
		hipFree(d_pairs);
		hipFree(d_parent_tnx);
		hipFree(d_child_tnx);
		hipFree(d_child_item);
		delete[] pairsGen;
	}	
	std::cout<<"total time: "<<totalTime<<" milliseconds--------------------------"<<std::endl;
	return 0;
}
int get_one(unsigned int value){
	int count = 0;
	unsigned int flag = 1;
	for(int i=0; i<8; i++) {
		if((value&flag) == flag)
			++count;
		value = value>>1;
	}
	return count;
}
